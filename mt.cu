#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>

void glaxpy(int n, float a, float *x, float *y)
{
  for (int i = 0; i < n; ++i)
      y[i] = a*x[i] + y[i];
}


__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipEvent_t start, stop;



  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  hipEventRecord(start);
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  hipEventRecord(stop);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);
  printf("Time: %f\n", milliseconds);
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);


// lets do cpu
  int NN = 1<<20;
  float *xx, *yy;
  xx = (float*)malloc(NN*sizeof(float));
  yy = (float*)malloc(NN*sizeof(float));

  for (int i = 0; i < NN; i++) {
    xx[i] = 1.0f;
    yy[i] = 2.0f;
  }


  hipEvent_t sttart, sttop;
  hipEventCreate(&sttart);
  hipEventCreate(&sttop);
  hipEventRecord(sttart);
  glaxpy(N, 2.0f, xx, yy);
  hipEventRecord(sttop);

  // hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  hipEventSynchronize(sttop);
  float miilliseconds = 0;
  hipEventElapsedTime(&miilliseconds, sttart, sttop);

    // float maxError = 0.0f;
    // for (int i = 0; i < N; i++)
    //   maxError = max(maxError, abs(y[i]-4.0f));
    // printf("Max error: %f\n", maxError);
  printf("Timecpp: %f\n", miilliseconds);
    // hipFree(d_x);
    // hipFree(d_y);
  free(xx);
  free(yy);
}
